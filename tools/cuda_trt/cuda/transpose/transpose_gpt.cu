#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "transpose.hpp"
#include "utils.hpp"

__global__ void transpose_matrix(const int* input_matrix, int* output_matrix, const int rows, const int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < rows && y < cols) {
        output_matrix[y * rows + x] = input_matrix[x * cols + y];
    }
}

void transpose_gpt()
{
    const int rows = 3;
    const int cols = 3;
    const int input_size = rows * cols;
    const int output_size = cols * rows;

    // 定义输入矩阵
    int input_matrix[rows][cols] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};

    // 定义输出矩阵
    int output_matrix[cols][rows];

    // 将输入矩阵和输出矩阵复制到GPU内存中
    int* d_input_matrix;
    int* d_output_matrix;
    hipMalloc((void**)&d_input_matrix, input_size * sizeof(int));
    hipMalloc((void**)&d_output_matrix, output_size * sizeof(int));
    hipMemcpy(d_input_matrix, input_matrix, input_size * sizeof(int), hipMemcpyHostToDevice);

    // 定义CUDA线程块和线程数
    dim3 threads_per_block(16, 16);
    dim3 blocks_per_grid((rows + threads_per_block.x - 1) / threads_per_block.x, (cols + threads_per_block.y - 1) / threads_per_block.y);

    // 调用CUDA函数
    transpose_matrix<<<blocks_per_grid, threads_per_block>>>(d_input_matrix, d_output_matrix, rows, cols);

    // 将结果从GPU内存中复制回CPU内存中
    CUDA_CHECK(hipMemcpy(output_matrix, d_output_matrix, output_size * sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipDeviceSynchronize());

    // 打印输入矩阵和输出矩阵
    std::cout << "Input Matrix:" << std::endl;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            std::cout << input_matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "Output Matrix:" << std::endl;
    for (int i = 0; i < cols; i++) {
        for (int j = 0; j < rows; j++) {
            std::cout << output_matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // 释放GPU内存
    hipFree(d_input_matrix);
    hipFree(d_output_matrix);
}


