
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (1024)
#define FULL_DATA_SIZE (N*500)

// nvcc -o 1-event 1-event.cu
// 类似cpu统计耗时一样，event的一个用法是可以统计出程序在gpu上运行的时长
// https://unofficial-sendoh.gitbook.io/unofficialsendoh/a/cuda-programming/cuda-events-and-streams

__global__ void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    c[tid] = a[tid] + b[tid];
}

int main(void) {
  float elapsedTime;
  hipEvent_t start, stop;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  int host_a[FULL_DATA_SIZE], host_b[FULL_DATA_SIZE], host_c[FULL_DATA_SIZE];
  int *dev_a, *dev_b, *dev_c;
  
  hipMalloc((void**)&dev_a, N * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  for (int i = 0; i < FULL_DATA_SIZE; i+=N) {
    hipMemcpy(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice);
    add<<<256,256>>>(dev_a, dev_b, dev_c);
    hipMemcpy(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  
  printf("Time taken: %3.1f ms\n", elapsedTime);
  
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}