
#include <hip/hip_runtime.h>
#include <stdio.h>

// https://unofficial-sendoh.gitbook.io/unofficialsendoh/a/cuda-programming/cuda-events-and-streams

#define N (1024)
#define FULL_DATA_SIZE (N*500)

__global__ void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    c[tid] = a[tid] + b[tid];
}

int main(void) {

  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  
  int *host_a, *host_b, *host_c;
  int *dev_a0, *dev_b0, *dev_c0;
  int *dev_a1, *dev_b1, *dev_c1;
  hipMalloc((void**)&dev_a0, N * sizeof(int));
  hipMalloc((void**)&dev_b0, N * sizeof(int));
  hipMalloc((void**)&dev_c0, N * sizeof(int));
  hipMalloc((void**)&dev_a1, N * sizeof(int));
  hipMalloc((void**)&dev_b1, N * sizeof(int));
  hipMalloc((void**)&dev_c1, N * sizeof(int));
  hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = rand();
    host_b[i] = rand();
  }

  for (int i = 0; i < FULL_DATA_SIZE; i+=2*N) {
    hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
    add<<<256,256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
    hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0); 
    hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
    add<<<256,256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
    hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1); 
  }

  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time taken: %3.1f ms\n", elapsedTime);
  hipFree(dev_a0);
  hipFree(dev_b0);
  hipFree(dev_c0);
  hipFree(dev_a1);
  hipFree(dev_b1);
  hipFree(dev_c1);
  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);
  hipStreamDestroy(stream0);
  hipStreamDestroy(stream1);
  return 0;
}