#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// 这段CUDA代码创建了两个CUDA流，分别在两个流中执行不同的计算任务。
// 在流1中，使用一个kernel函数将deviceData数组中的每个元素都设置为42+1。
// 在流2中，使用hipMemcpyAsync函数将deviceData数组中的数据异步地拷贝到hostData数组中。
// 最后，主机等待流1和流2的任务完成，然后打印hostData数组中的数据，并释放所有的资源。
// 这个程序的目的是展示如何使用CUDA流来并行执行多个计算任务。

#define N 1000

__global__ void kernel(int* data, int value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        data[tid] = value + 1;
    }
}

int main() {
    int* hostData;
    int* deviceData;
    hipStream_t stream1, stream2;

    // 分配主机内存和设备内存
    hostData = (int*)malloc(N * sizeof(int));
    hipMalloc((void**)&deviceData, N * sizeof(int));

    // 创建cuda流
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // 在流1中执行计算
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    kernel << <grid, block, 0, stream1 >> > (deviceData, 42);

    // 在流2中执行计算
    hipMemcpyAsync(hostData, deviceData, N * sizeof(int), hipMemcpyDeviceToHost, stream2);

    // 主机等待流1和流2的任务完成
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // 打印结果
    for (int i = 0; i < N; i++) {
        printf("%d ", hostData[i]);
    }
    printf("\n");

    // 释放资源
    free(hostData);
    hipFree(deviceData);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}